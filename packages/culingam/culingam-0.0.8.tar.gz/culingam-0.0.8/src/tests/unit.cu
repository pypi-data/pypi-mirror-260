#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <iostream>
#include "basic.cuh"


extern "C" void run_compute_mean(double *data, double *mean, int size, int threads_i) {
    double *d_data, *d_mean;
    hipMalloc(&d_data, size * sizeof(double));
    hipMalloc(&d_mean, sizeof(double));

    hipMemcpy(d_data, data, size * sizeof(double), hipMemcpyHostToDevice);

    int threads = threads_i;
    int blocks = (size + threads - 1) / threads;
    int sharedSize = threads * sizeof(double);

    compute_mean<<<blocks, threads, sharedSize>>>(d_data, d_mean, size);

    hipMemcpy(mean, d_mean, sizeof(double), hipMemcpyDeviceToHost);
    *mean /= size;

    hipFree(d_data);
    hipFree(d_mean);
}


extern "C" void run_compute_covariance_variance(double *xi, double *xj, double *mean_xi, double *mean_xj, double *covariance, double *variance, int size, int blockSize) {
    double *d_xi, *d_xj, *d_mean_xi, *d_mean_xj, *d_covariance, *d_variance;

    // Allocate and copy memory to device
    hipMalloc(&d_xi, size * sizeof(double));
    hipMalloc(&d_xj, size * sizeof(double));
    hipMalloc(&d_mean_xi, sizeof(double));
    hipMalloc(&d_mean_xj, sizeof(double));
    hipMalloc(&d_covariance, sizeof(double));
    hipMalloc(&d_variance, sizeof(double));

    hipMemcpy(d_xi, xi, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_xj, xj, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_mean_xi, mean_xi, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_mean_xj, mean_xj, sizeof(double), hipMemcpyHostToDevice);

    dim3 block(blockSize);
    dim3 grid((size + block.x - 1) / block.x);
    size_t sharedSize = 2 * block.x * sizeof(double);

    compute_covariance_variance<<<grid, block, sharedSize>>>(d_xi, d_xj, d_mean_xi, d_mean_xj, d_covariance, d_variance, size);

    hipMemcpy(covariance, d_covariance, sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(variance, d_variance, sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_xi);
    hipFree(d_xj);
    hipFree(d_mean_xi);
    hipFree(d_mean_xj);
    hipFree(d_covariance);
    hipFree(d_variance);
}


extern "C" void run_element_wise_division(double *r, double *constant_std, double *result, int n, int blockSize) {
    double *d_r, *d_constant_std, *d_result;

    // Allocate and copy memory to device
    hipMalloc(&d_r, n * sizeof(double));
    hipMalloc(&d_constant_std, sizeof(double));
    hipMalloc(&d_result, n * sizeof(double));

    hipMemcpy(d_r, r, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_constant_std, constant_std, sizeof(double), hipMemcpyHostToDevice);

    dim3 block(blockSize);
    dim3 grid((n + block.x - 1) / block.x);

    element_wise_division<<<grid, block>>>(d_r, d_constant_std, d_result, n);

    hipMemcpy(result, d_result, n * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_r);
    hipFree(d_constant_std);
    hipFree(d_result);
}



extern "C" void run_compute_residual(double *xi, double *xj, double *scaling_factor, double *residual, int size, int blockSize) {
    double *d_xi, *d_xj, *d_scaling_factor, *d_residual;

    // Allocate and copy memory to device
    hipMalloc(&d_xi, size * sizeof(double));
    hipMalloc(&d_xj, size * sizeof(double));
    hipMalloc(&d_scaling_factor, sizeof(double));
    hipMalloc(&d_residual, size * sizeof(double));

    hipMemcpy(d_xi, xi, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_xj, xj, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_scaling_factor, scaling_factor, sizeof(double), hipMemcpyHostToDevice);

    dim3 block(blockSize);
    dim3 grid((size + block.x - 1) / block.x);

    compute_residual<<<grid, block>>>(d_xi, d_xj, d_scaling_factor, d_residual, size);

    hipMemcpy(residual, d_residual, size * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_xi);
    hipFree(d_xj);
    hipFree(d_scaling_factor);
    hipFree(d_residual);
}





extern "C" void run_compute_std(double *A, double *mean, double *std, int size, int blockSize) {
    double *d_A, *d_mean, *d_std;

    // Allocate and copy memory to device
    hipMalloc(&d_A, size * sizeof(double));
    hipMalloc(&d_mean, sizeof(double));
    hipMalloc(&d_std, sizeof(double));

    hipMemcpy(d_A, A, size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_mean, mean, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_std, std, sizeof(double), hipMemcpyHostToDevice);

    dim3 block(blockSize);
    dim3 grid((size + block.x - 1) / block.x);
    size_t sharedSize = block.x * sizeof(double);

    compute_std<<<grid, block, sharedSize>>>(d_A, d_mean, d_std, size);

    hipMemcpy(std, d_std, sizeof(double), hipMemcpyDeviceToHost);
    *std /= size;
    *std = sqrtf(*std);

    // Free memory
    hipFree(d_A);
    hipFree(d_mean);
    hipFree(d_std);
}


extern "C" void run_calculate_statistics(double *A, int m, int n, double *means, double *stds, int blockSize) {
    double *d_A, *d_means, *d_stds;

    // Allocate and copy memory to device
    hipMalloc(&d_A, m * n * sizeof(double));
    hipMalloc(&d_means, n * sizeof(double));
    hipMalloc(&d_stds, n * sizeof(double));

    hipMemcpy(d_A, A, m * n * sizeof(double), hipMemcpyHostToDevice);

    dim3 block(blockSize);
    dim3 grid((n + block.x - 1) / block.x);

    calculate_statistics<<<grid, block>>>(d_A, m, n, d_means, d_stds);

    hipMemcpy(means, d_means, n * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(stds, d_stds, n * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A);
    hipFree(d_means);
    hipFree(d_stds);
}



extern "C" void run_standardize_column(double *A, int m, int n, double *means, double *stds, int threads_i) {
    double *d_A;
    double *d_means, *d_stds;

    // Allocate and copy memory to device
    hipMalloc(&d_A, m * n * sizeof(double));
    hipMalloc(&d_means, n * sizeof(double));
    hipMalloc(&d_stds, n * sizeof(double));

    hipMemcpy(d_A, A, m * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_means, means, n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_stds, stds, n * sizeof(double), hipMemcpyHostToDevice);

    dim3 block(threads_i, threads_i);
    int numChunks = (m + threads_i - 1) / threads_i;

    for (int chunk = 0; chunk < numChunks; ++chunk) {
        int startRow = chunk *  threads_i;
        dim3 grid((threads_i + block.x - 1) / block.x,
                         (n + block.y - 1) / block.y);
        standardize_column<<<grid, block>>>(d_A, m, n, d_means, d_stds, startRow);
    }

    hipMemcpy(A, d_A, m * n * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_A);
    hipFree(d_means);
    hipFree(d_stds);
}



extern "C" void run_compute_log_cosh(double *u, double *log_cosh_sum, int size, int blockSize) {
    double *d_u, *d_log_cosh_sum;

    hipMalloc(&d_u, size * sizeof(double));
    hipMalloc(&d_log_cosh_sum, sizeof(double));

    hipMemcpy(d_u, u, size * sizeof(double), hipMemcpyHostToDevice);

    dim3 block(blockSize);
    dim3 grid((size + block.x - 1) / block.x);
    size_t sharedSize = block.x * sizeof(double);

    compute_log_cosh<<<grid, block, sharedSize>>>(d_u, d_log_cosh_sum, size);

    hipMemcpy(log_cosh_sum, d_log_cosh_sum, sizeof(double), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_u);
    hipFree(d_log_cosh_sum);
}



extern "C" void run_compute_u_exp(double *u, double *u_exp_sum, int size, int blockSize) {
    double *d_u, *d_u_exp_sum;

    hipMalloc(&d_u, size * sizeof(double));
    hipMalloc(&d_u_exp_sum, sizeof(double));

    hipMemcpy(d_u, u, size * sizeof(double), hipMemcpyHostToDevice);

    dim3 block(blockSize);
    dim3 grid((size + block.x - 1) / block.x);
    size_t sharedSize = block.x * sizeof(double);

    compute_u_exp<<<grid, block, sharedSize>>>(d_u, d_u_exp_sum, size);

    hipMemcpy(u_exp_sum, d_u_exp_sum, sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_u);
    hipFree(d_u_exp_sum);
}



extern "C" void run_process_column(double *X, double *column, int m, int n, int col_idx, int threads_i) {
    double *d_column;
    double *d_X;

    hipMalloc(&d_X, m * n * sizeof(double));
    hipMemcpy(d_X, X, m * n  * sizeof(double), hipMemcpyHostToDevice);
    hipMalloc(&d_column, m * sizeof(double));

    int blocks = (m + threads_i - 1) / threads_i;
    process_column<<<blocks, threads_i>>>(d_X, d_column, m, n, col_idx);

    hipMemcpy(column, d_column, m * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_column);
}



extern "C" void end_end_residual(double *data, int M, int N, int m, int *U, int uN)
{
    double *d_X;

    hipMalloc(&d_X, M * N * sizeof(double));
    hipMemcpy(d_X, data, M * N * sizeof(double), hipMemcpyHostToDevice);

    dim3 blocks(256);
    dim3 grid((M + blocks.x - 1) / blocks.x);

    size_t sharedMemSize = blocks.x * sizeof(double);

    double *xj, *xm;

    hipMalloc(&xj, M * sizeof(double));
    hipMalloc(&xm, M * sizeof(double));

    double *means_xj, *means_xm;

    hipMalloc(&means_xj, sizeof(double));
    hipMalloc(&means_xm, sizeof(double));

    double *h_means_xj, *h_means_xm;

    hipHostMalloc(&h_means_xj, sizeof(double));
    hipHostMalloc(&h_means_xm, sizeof(double));

    double *d_residual_ij, *d_covariance_i, *d_variance_j, *scaling_factor_ij;

    hipMalloc(&d_residual_ij, M * sizeof(double));

    hipMalloc(&d_covariance_i, sizeof(double));
    hipMalloc(&d_variance_j, sizeof(double));
    hipMalloc(&scaling_factor_ij, sizeof(double));

    int j = 0;
    for (int uj = 0; uj < uN; ++uj)
    {
        j = U[uj];
        if (j != m)
        {
            process_column<<<grid, blocks>>>(d_X, xj, M, N, j);
            process_column<<<grid, blocks>>>(d_X, xm, M, N, m);

            compute_mean<<<grid, blocks, sharedMemSize>>>(xj, means_xj, M);
            compute_mean<<<grid, blocks, sharedMemSize>>>(xm, means_xm, M);

            divonhost(h_means_xj, means_xj, M);
            divonhost(h_means_xm, means_xm, M);

            compute_covariance_variance<<<grid, blocks, 2 * sharedMemSize>>>(xj, xm, means_xj, means_xm, d_covariance_i, d_variance_j, M);
            element_wise_division<<<grid, blocks>>>(d_covariance_i, d_variance_j, scaling_factor_ij, 1);

            compute_residual<<<grid, blocks>>>(xj, xm, scaling_factor_ij, d_residual_ij, M);
            update(d_X, d_residual_ij, M, N, j);

        }
    }

    hipMemcpy(data, d_X, M * N * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(xj);
    hipFree(xm);
    hipFree(means_xj);
    hipFree(means_xm);
    hipFree(d_residual_ij);
    hipFree(d_covariance_i);
    hipFree(d_variance_j);
    hipFree(scaling_factor_ij);
    hipHostFree(h_means_xj);
    hipHostFree(h_means_xm);
    hipFree(d_X);

}
